/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclAllReduceFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  size_t nbytes = count*ncclTypeSize(datatype);
  fprintf(stderr, "Before opcount\n");
  INFO(COLL,"opCount %lx sendbuff %p recvbuff %p count %zi size %zi datatype %d op %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, nbytes, datatype, op, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    fprintf(stderr, "befor save proxies\n");
    NCCLCHECK(transportSaveProxies(ALLREDUCE_SUBSTEPS, ALLREDUCE_BUFCHUNKS, (comm->nRanks)*2-2, comm->nRanks, nbytes, proxyPatternRing, comm));
    fprintf(stderr, "after save proxies\n");
    NCCLCHECK(saveKernel(ncclCollAllReduce, sendbuff, recvbuff, count, datatype, op, root, comm, stream, nbytes, comm->nRanks));
    //NCCLCHECK(transportSaveProxies(ALLREDUCE_SUBSTEPS, ALLREDUCE_BUFCHUNKS, (comm->nRanks)*2-2, comm->nRanks, nbytes, proxyPatternRing, comm));
    // NCCLCHECK(saveKernel(ncclCollAllReduce, sendbuff, recvbuff, count, datatype, op, root, comm, stream, nbytes, comm->nRanks));

  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  fprintf(stderr, "Enqueue check\n");
  return ncclEnqueueCheck(ncclAllReduceFunc, "AllReduce", sendbuff, recvbuff, count, datatype,
          op, 0, comm->nodeComm, stream);
}
