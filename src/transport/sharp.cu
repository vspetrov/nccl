#include "core.h"
#include "transport.h"
#include "nvmlwrap.h"
#include "net.h"
#include "param.h"
#include "nvlink.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <mpi.h>
#include <bootstrap.h>
#include "sharp/api/version.h"
#include "sharp/api/sharp_coll.h"

extern void* sharpBootstrapCtx;

struct sharpSendResources {
  void* netSendComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclSendMem* hostDevMem;
  int netDev;
  bool cudaSupport;
  struct ncclRecvMem* devNetMem;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

struct sharpRecvResources {
  void* netListenComm;
  void* netRecvComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclRecvMem* hostDevMem;
  int netDev;
  bool cudaSupport;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

ncclResult_t sharpSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct sharpSendResources* resources;
  NCCLCHECK(ncclCalloc(&resources, 1));
  ring->sharp.transportResources = resources;

  fprintf(stderr,"Sharp setup\n");
  #if 0
  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->scores);
  resources->cudaSupport = false;
#endif

  int cudaDev;
  CUDACHECK(hipGetDevice(&cudaDev));
  resources->cudaSupport = true;

  int size = offsetof(struct ncclRecvMem, buff)+ring->buffSize;
  if (resources->cudaSupport) {
    NCCLCHECK(ncclCudaCalloc((char**)(&resources->devNetMem), size));
  }

  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostRecvMem, (void**)&resources->devHostRecvMem, size));
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostSendMem, (void**)&resources->devHostSendMem, size));
  MPI_Comm_split(MPI_COMM_WORLD, ring->sharpNodeRank, ring->sharpNodeRank, &(ring->mpiNodeComm));
  //  sharpBootstrapCtx = commState;
  struct sharp_coll_comm_init_spec comm_spec;
  comm_spec.rank      = ring->sharpNodeRank;
  comm_spec.size      = ring->sharpCommSize;
  #if 0
  uint32_t *gwr = NULL;
#if SHARP_API > SHARP_VERSION(1,4)
  gwr = (uint32_t*)malloc(nranks*sizeof(uint32_t));
  gwr[rank] = main_comm->rank;
  NCCLCHECK(bootstrapAllGather(commState, gwr, sizeof(uint32_t)));
  comm_spec.group_world_ranks = gwr;
#endif
  comm_spec.is_comm_world = 0;
  comm_spec.oob_ctx   = commState;
  int ret = sharp_coll_comm_init(main_comm->sharpCtx, &comm_spec, (struct sharp_coll_comm **)&comm->sharpComm);
  if (gwr) free(gwr);
  if (ret < 0) {
      fprintf(stderr, "sharp group create failed:%s(%d)\n", sharp_coll_strerror(ret), ret);
      return ncclInternalError;
  } else {
      fprintf(stderr, "SHARP GROUP CREATE SUCCESS, %p\n", comm->sharpComm);
  }
  #endif
  return ncclSuccess;
}


ncclResult_t sharpProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct sharpSendResources* resources = (struct sharpSendResources*) (ring->sharp.transportResources);
  const int llMode = args->llMode;
  volatile uint64_t* prevTail = &resources->hostRecvMem->tail;

  struct ncclSendMem* prevMem = resources->hostDevMem ? resources->hostDevMem : resources->hostSendMem;
  volatile uint64_t* prevHead = llMode ? &prevMem->llHead : &prevMem->head;
  struct ncclRecvMem* localMem = resources->cudaSupport ? resources->devNetMem : resources->hostRecvMem;
  char* localBuff = llMode ? resources->hostRecvMem->llBuff : localMem->buff;
  volatile int* sizesFifo = llMode ? resources->hostRecvMem->llSizesFifo : resources->hostRecvMem->sizesFifo;
  int buffSize = llMode ? NCCL_LL_BUFF_SIZE : ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  while (!(*prevHead)){ 
    ;;
  }
  int rank, lrank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_rank(ring->mpiNodeComm, &lrank);
  int offset, sizeReduce;
  offset = sizesFifo[0];
  //sizeReduce = sizesFifo[1];
  sizeReduce = 8;
  
  #if 1
  for(int k = 0; k<4;k++){
    if (rank == k){
      fprintf(stderr, "before allreduce Thread - gRank %d lrank %d: ", rank, lrank);
      for(int l = 0; l < 8; l++)
	fprintf(stderr, "%f ", ((float*)ring->recv.conn.buff)[l+offset]);
      fprintf(stderr, "\n");
    }
    //     MPI_Barrier(MPI_COMM_WORLD);
  }
  #endif
  //  if (rank == 0)
  fprintf(stderr, "grank %d lrank %d: Offset = %d size = %d\n", rank, lrank, offset, sizeReduce);
  MPI_Barrier(MPI_COMM_WORLD);
  //  volatile float* redBuf = (float*)ring->recv.conn.buff;
  float *redBuf = (float*)ring->recv.conn.buff;
  MPI_Allreduce(MPI_IN_PLACE, (float*)redBuf+offset, sizeReduce, MPI_FLOAT, MPI_SUM, ring->mpiNodeComm);
  //  MPI_Barrier(MPI_COMM_WORLD);
  __sync_synchronize();
  #if 0
  for(int k = 0; k<4;k++){
    if (rank == k){
      fprintf(stderr, "after allreduce Thread - grank %d lrank %d: ", rank, lrank);
      for(int l = 0; l < 8; l++)
	fprintf(stderr, "%f ", ((float*)ring->recv.conn.buff)[l+offset]);
      fprintf(stderr, "\n");
    }
    //  MPI_Barrier(MPI_COMM_WORLD);
  }
  #endif
  fprintf(stderr,"Hello world from rank %d!!\n", rank);
  ++(*prevTail);  
  return ncclSuccess;
}

ncclResult_t sharpConnect(struct ncclConnect* connectInfo, struct ncclConnector* send) {
  // Setup device pointers
  struct sharpSendResources* resources = (struct sharpSendResources*)send->transportResources;

  if (resources->cudaSupport) {
    send->conn.buff = resources->devNetMem->buff;
    // We don't use devMem for llMode because the CPU has to read the data
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  } else {
    send->conn.buff = resources->devHostRecvMem->buff;
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  }
  send->conn.tail = &resources->devHostRecvMem->tail;
  send->conn.opCount = &resources->devHostRecvMem->opCount;
  send->conn.fifo = resources->devHostRecvMem->sizesFifo;
  send->conn.llFifo = resources->devHostRecvMem->llSizesFifo;

  if (resources->hostDevMem == NULL) {
    send->conn.head = &resources->devHostSendMem->head;
    send->conn.llHead = &resources->devHostSendMem->llHead;
  }

  //Sharp comm init should be here.
  return ncclSuccess;
}

struct ncclTransport sharpTransport = {
				       "SRP",
				       NULL,
				       NULL,
				       NULL,
				       {sharpSetup, sharpConnect, NULL, sharpProxy},
				       {sharpSetup, sharpConnect, NULL, sharpProxy}
};
